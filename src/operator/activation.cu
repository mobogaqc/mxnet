#include "hip/hip_runtime.h"
/*!
 * Copyright (c) 2015 by Contributors
 * \file activation.cu
 * \brief
 * \author Bing Xu
*/

#include "./activation-inl.h"
#include "./mshadow_op.h"

namespace mxnet {
namespace op {
template<>
Operator *CreateActivationOp<gpu>(ActivationOpType type) {
  switch(type) {
    case kReLU: return new ActivationOp<gpu, mshadow_op::relu, mshadow_op::relu_grad>();
    case kSigmoid: return new ActivationOp<gpu, mshadow_op::sigmoid, mshadow_op::sigmoid_grad>();
    case kTanh: return new ActivationOp<gpu, mshadow_op::tanh, mshadow_op::tanh_grad>();
    default: return NULL;
  }
}
}  // op
}  // namespace mxnet

