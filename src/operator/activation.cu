#include "hip/hip_runtime.h"
/*!
 * Copyright (c) 2015 by Contributors
 * \file activation.cu
 * \brief
 * \author Bing Xu
*/
#include "./activation-inl.h"
#include "./mshadow_op.h"

namespace mxnet {
namespace op {
template<>
Operator *CreateOp<gpu>(ActivationParam param) {
  switch(param.act_type) {
    case kReLU: return new ActivationOp<gpu, mshadow_op::relu, mshadow_op::relu_grad>();
    case kSigmoid: return new ActivationOp<gpu, mshadow_op::sigmoid, mshadow_op::sigmoid_grad>();
    case kTanh: return new ActivationOp<gpu, mshadow_op::tanh, mshadow_op::tanh_grad>();
    default:
      LOG(FATAL) << "unknown activation";
      return NULL;
  }
}
}  // op
}  // namespace mxnet

