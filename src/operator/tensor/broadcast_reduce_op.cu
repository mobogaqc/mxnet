#include "hip/hip_runtime.h"
/*!
 *  Copyright (c) 2016 by Contributors
 * \file broadcast_reduce_op.cu
 * \brief GPU Implementation of broadcast and reduce functions.
 */
#include "./broadcast_reduce_op.h"

namespace mxnet {
namespace op {
NNVM_REGISTER_OP(sum)
.set_attr<FCompute>("FCompute<gpu>", ReduceAxesCompute<gpu, mshadow::red::sum>);

NNVM_REGISTER_OP(_backward_sum)
.set_attr<FCompute>("FCompute<gpu>", ReduceAxesBackwardUseNone<gpu>);

NNVM_REGISTER_OP(max)
.set_attr<FCompute>("FCompute<gpu>", ReduceAxesCompute<gpu, mshadow::red::maximum>);

NNVM_REGISTER_OP(_backward_max)
.set_attr<FCompute>("FCompute<gpu>", ReduceAxesBackwardUseInOut<gpu, mshadow_op::eq>);

NNVM_REGISTER_OP(min)
.set_attr<FCompute>("FCompute<gpu>", ReduceAxesCompute<gpu, mshadow::red::minimum>);

NNVM_REGISTER_OP(_backward_min)
.set_attr<FCompute>("FCompute<gpu>", ReduceAxesBackwardUseInOut<gpu, mshadow_op::eq>);

NNVM_REGISTER_OP(broadcast_axis)
.set_attr<FCompute>("FCompute<gpu>", BroadcastCompute<gpu>);

NNVM_REGISTER_OP(broadcast_to)
.set_attr<FCompute>("FCompute<gpu>", BroadcastCompute<gpu>);

NNVM_REGISTER_OP(_broadcast_backward)
.set_attr<FCompute>("FCompute<gpu>", ReduceAxesCompute<gpu, mshadow::red::sum>);

MXNET_OPERATOR_REGISTER_REDUCE_AXIS(argmax)
.set_attr<FCompute>("FCompute<gpu>", SearchAxisCompute<gpu, mshadow::red::maximum>);

MXNET_OPERATOR_REGISTER_REDUCE_AXIS(argmin)
.set_attr<FCompute>("FCompute<gpu>", SearchAxisCompute<gpu, mshadow::red::minimum>);

// Legacy support
NNVM_REGISTER_OP(argmax_channel)
.set_attr<FCompute>("FCompute<gpu>", SearchAxisCompute<gpu, mshadow::red::maximum>);

NNVM_REGISTER_OP(norm)
.set_attr<FCompute>("FCompute<gpu>", L2NormCompute<gpu>);

}  // namespace op
}  // namespace mxnet
