#include "hip/hip_runtime.h"
/*!
 *  Copyright (c) 2016 by Contributors
 * \file elemwise_unary_op.cu
 * \brief GPU Implementation of unary function.
 */
#include "./elemwise_unary_op.h"
#include "./elemwise_binary_op.h"

namespace mxnet {
namespace op {
// copy
NNVM_REGISTER_OP(_copy)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::identity>);

// negative
NNVM_REGISTER_OP(negative)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::negation>);

// abs
NNVM_REGISTER_OP(abs)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::abs>);

NNVM_REGISTER_OP(_backward_abs)
.set_attr<FCompute>("FCompute<gpu>", BinaryCompute<gpu, unary_bwd<mshadow_op::sign> >);

// sign
NNVM_REGISTER_OP(sign)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::sign>);

NNVM_REGISTER_OP(_backward_sign)
.set_attr<FCompute>("FCompute<gpu>", BinaryCompute<gpu, unary_bwd<mshadow_op::sign_grad> >);

// round
NNVM_REGISTER_OP(round)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::round>);

// ceil
NNVM_REGISTER_OP(ceil)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::ceil>);

// floor
NNVM_REGISTER_OP(floor)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::floor>);

// square
NNVM_REGISTER_OP(square)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::square>);

NNVM_REGISTER_OP(_backward_square)
.set_attr<FCompute>("FCompute<gpu>", BinaryCompute<gpu, unary_bwd<mshadow_op::square_grad> >);

// sqrt
NNVM_REGISTER_OP(sqrt)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::square_root>);

NNVM_REGISTER_OP(_backward_sqrt)
.set_attr<FCompute>("FCompute<gpu>", BinaryCompute<gpu, unary_bwd<mshadow_op::square_root_grad> >);

// rsqrt
NNVM_REGISTER_OP(rsqrt)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::reciprocal_square_root>);

NNVM_REGISTER_OP(_backward_rsqrt)
.set_attr<FCompute>("FCompute<gpu>",
  BinaryCompute<gpu, unary_bwd<mshadow_op::reciprocal_square_root_grad> >);

// exp
NNVM_REGISTER_OP(exp)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::exp>);

// log
NNVM_REGISTER_OP(log)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::log>);

NNVM_REGISTER_OP(_backward_log)
.set_attr<FCompute>("FCompute<gpu>", BinaryCompute<gpu, unary_bwd<mshadow_op::log_grad> >);

// cos
NNVM_REGISTER_OP(cos)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::cos>);

NNVM_REGISTER_OP(_backward_cos)
.set_attr<FCompute>("FCompute<gpu>", BinaryCompute<gpu, unary_bwd<mshadow_op::cos_grad> >);

// sin
NNVM_REGISTER_OP(sin)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::sin>);

NNVM_REGISTER_OP(_backward_sin)
.set_attr<FCompute>("FCompute<gpu>", BinaryCompute<gpu, unary_bwd<mshadow_op::sin_grad> >);

}  // namespace op
}  // namespace mxnet
